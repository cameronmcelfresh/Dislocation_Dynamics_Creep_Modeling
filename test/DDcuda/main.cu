#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc main.cu -o DDcuda
// ln -s /usr/local/cuda/include/crt/math_functions.hpp /usr/local/cuda/include/math_functions.hpp
// https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
// https://stackoverflow.com/questions/9985912/how-do-i-choose-grid-and-block-dimensions-for-cuda-kernels
// The number of threads per block should be a round multiple of the warp size, which is 32 on all current hardware.
// reference for hipDeviceProp_t
// https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html

#include <chrono>
#include <iomanip>
#include <omp.h>


#include <DDcuda.cpp>


int main(void)
{

    model::DDcuda ddc;

std::vector<Eigen::Vector3d> x;

 std::vector<double> y;
std::vector<double> z;

const int N=1e8;
x.reserve(N);
y.reserve(N);
z.reserve(N);

for (size_t n=0;n<N;++n)
{
x.push_back(Eigen::Vector3d::Random().normalized());
y.push_back(0);
z.push_back(0);

}

// CUDA code
std::cout<<"CUDA"<<std::endl;
const auto t0= std::chrono::system_clock::now();
ddc.computeStressAtQuadrature(x,y);
std::cout<<std::setprecision(3)<<std::scientific<<" ["<<(std::chrono::duration<double>(std::chrono::system_clock::now()-t0)).count()<<" sec]."<<std::endl;

// serial code
std::cout<<"serial"<<std::endl;
const auto t1= std::chrono::system_clock::now();
for (size_t n=0;n<N;++n)
{
    z[n]=x[n].dot(x[n]);
}
std::cout<<std::setprecision(3)<<std::scientific<<" ["<<(std::chrono::duration<double>(std::chrono::system_clock::now()-t1)).count()<<" sec]."<<std::endl;

// omp code
//std::cout<<"serial"<<std::endl;
//const auto t2= std::chrono::system_clock::now();
//#pragma omp parallel for
//for (int n=0;n<10000000;++n)
//{
//z[n]=x[n].dot(x[n]);
//}
//std::cout<<std::setprecision(3)<<std::scientific<<" ["<<(std::chrono::duration<double>(std::chrono::system_clock::now()-t2)).count()<<" sec]."<<std::endl;


  return 0;
}
