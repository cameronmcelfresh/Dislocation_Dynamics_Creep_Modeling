#include "hip/hip_runtime.h"
// /usr/local/cuda/bin/nvcc main.cu -o add_cuda
// ln -s /usr/local/cuda/include/crt/math_functions.hpp /usr/local/cuda/include/math_functions.hpp
// https://devblogs.nvidia.com/easy-introduction-cuda-c-and-c/
// https://stackoverflow.com/questions/9985912/how-do-i-choose-grid-and-block-dimensions-for-cuda-kernels
// The number of threads per block should be a round multiple of the warp size, which is 32 on all current hardware.
// reference for hipDeviceProp_t
// https://docs.nvidia.com/cuda/cuda-runtime-api/structcudaDeviceProp.html
#include <iostream>
#include <math.h>

#include <Eigen/Dense>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

int getSPcores(hipDeviceProp_t devProp)
{
int cores = 0;
int mp = devProp.multiProcessorCount;
switch (devProp.major){
case 2: // Fermi
if (devProp.minor == 1) cores = mp * 48;
else cores = mp * 32;
break;
case 3: // Kepler
cores = mp * 192;
break;
case 5: // Maxwell
cores = mp * 128;
break;
case 6: // Pascal
if (devProp.minor == 1) cores = mp * 128;
else if (devProp.minor == 0) cores = mp * 64;
else printf("Unknown device type\n");
break;
case 7: // Volta
if (devProp.minor == 0) cores = mp * 64;
else printf("Unknown device type\n");
break;
default:
printf("Unknown device type\n");
break;
}
return cores;
}

int main(void)
{

    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
    prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
    prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
    2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

    printf("  # Warp size: %d\n",prop.warpSize);

    printf("  # cores: %d\n",getSPcores(prop));

    printf("  # streaming processor (SP) units: %d\n",prop.multiProcessorCount);

    printf("  # max thread per block: %d\n",prop.maxThreadsPerBlock);

    printf("  # max thread per SP: %d\n",prop.maxThreadsPerMultiProcessor);


}



  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}
