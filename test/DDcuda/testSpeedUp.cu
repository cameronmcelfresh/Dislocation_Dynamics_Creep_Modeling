
// /usr/local/cuda/bin/nvcc testSpeedUp.cu -o test -O3

// WARNING for OPTIMIZATION
// warning: compiling with nvcc -O3 filename.cu will pass the -O3 option to host code only.
// nvcc -Xptxas -O3,-v filename.cu
// https://stackoverflow.com/questions/43706755/how-can-i-get-the-nvcc-cuda-compiler-to-optimize-more

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>

#define N (1024*1024)
#define M (10000)
#define THREADS_PER_BLOCK 1024

void serial_add(double *a, double *b, double *c, int n, int m)
{
for(int index=0;index<n;index++)
{
for(int j=0;j<m;j++)
{
c[index] = a[index]*a[index] + b[index]*b[index];
}
}
}

__global__ void vector_add(double *a, double *b, double *c)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;
for(int j=0;j<M;j++)
{
c[index] = a[index]*a[index] + b[index]*b[index];
}
}

int main()
{
clock_t start,end;

double *a, *b, *c;
int size = N * sizeof( double );

a = (double *)malloc( size );
b = (double *)malloc( size );
c = (double *)malloc( size );

for( int i = 0; i < N; i++ )
{
a[i] = b[i] = i;
c[i] = 0;
}

start = clock();
serial_add(a, b, c, N, M);

printf( "c[0] = %d\n",0,c[0] );
printf( "c[%d] = %d\n",N-1, c[N-1] );

end = clock();

float time1 = ((float)(end-start))/CLOCKS_PER_SEC;
printf("Serial: %f seconds\n",time1);

start = clock();
double *d_a, *d_b, *d_c;


hipMalloc( (void **) &d_a, size );
hipMalloc( (void **) &d_b, size );
hipMalloc( (void **) &d_c, size );


hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


printf( "c[0] = %d\n",0,c[0] );
printf( "c[%d] = %d\n",N-1, c[N-1] );


free(a);
free(b);
free(c);
hipFree( d_a );
hipFree( d_b );
hipFree( d_c );

end = clock();
float time2 = ((float)(end-start))/CLOCKS_PER_SEC;
printf("CUDA: %f seconds, Speedup: %f\n",time2, time1/time2);

return 0;
}
